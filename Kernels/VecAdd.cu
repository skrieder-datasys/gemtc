
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ void VecAdd ( void* param1)
{
   float* mem = (float*)param1;
   int size = (int)mem[0];
   float* A = mem+1;
   float* B = A+size;
   float* C = B+size;
   int warp_size = 32;
   int tid = threadIdx.x;
   //printf("%d,%d\n",size, threadIdx.x);
   //C[tid] = A[tid] + B[tid];
   while (tid < size)
   {
      C[tid] = A[tid] + B[tid];
      //printf("%d=%d\n", tid, C[tid]);
      tid = tid + warp_size;
   }

#if 0 
   int* A = (int*)param1;
   int* B = (int*)param1;
   int* C = (int*)param1;

   int tid = threadIdx.x + blockIdx.x*blockDim.x;

   while (tid < 32) {
      C[tid] = A[tid] + B[tid];
      tid += blockDim.x*gridDim.x;
   }
#endif
}
