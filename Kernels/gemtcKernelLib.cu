
#include <hip/hip_runtime.h>
#include <stdio.h>

//This function will return the threadID from 0..31
//This ID is unique within the warp.
//Different from CUDA's ID which is unique within the block
__device__ int gemtcThreadID()
{ 
  return threadIdx.x % 32;
}

//This function will return the warpID of the calling thread
//This number will be unique within the block
__device__ int gemtcWarpID()
{ 
  return threadIdx.x / 32;
}

//SHARED MEMORY MANAGEMENT
__device__ void *SHARED_MEMORY;
__device__ int SHARED_SIZE;
__device__ int WARPS_PER_SM;

__device__ void gemtcInitSharedMemory(void *shared_mem, int mem_size, int warps)
{
  //This needs to initialize the three variables above
  //We need to statically have a large block on Shared Memory
  //  allocated when the program launches to divide
  SHARED_MEMORY = shared_mem;
  SHARED_SIZE = mem_size/warps;
  WARPS_PER_SM = warps;
}

__device__ void *gemtcSharedMemory()
{ 
  return ((char *)SHARED_MEMORY) + SHARED_SIZE*gemtcWarpID()/WARPS_PER_SM;
}
